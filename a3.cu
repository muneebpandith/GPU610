#include "hip/hip_runtime.h"
#define MAXSIZE 250000




#include <iostream>
#include <string>
#include <fstream>	//Writing to files
#include <chrono>	//Keep track of time
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
// to remove intellisense highlighting
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <algorithm>
#include ""

using namespace std::chrono;

int data[MAXSIZE];
//Main CUDA kernel implementing Sieve of Eratosthenes
__global__ static void CUDASieve(int *num, int range, int bNum, int tNum){
	const int threadId = threadIdx.x;
	const int blockId = blockIdx.x;
	int tmp = blockId*tNum + threadId;
	while (tmp < range){
		int i = 1;
		while (((2 * tmp + 3)*i + tmp + 1) < MAXSIZE){
			num[(2 * tmp + 3)*i + tmp + 1] = 0;
			i++;
		}
		tmp += bNum * tNum;
	}
}
void CUDAFilter(int *number, int size){
	for (int i = 0; i<size; i++)
		number[i] = 2 * i + 1;
	number[0] = 2;
}

void reportTime(const char* msg, steady_clock::duration span) {
	auto ms = duration_cast<milliseconds>(span);
	std::cout << msg << ms.count() << " millisecs" << std::endl;
}

void CPUgenPrime(uint64_t range, bool mode, std::ofstream &fileOut) {
	//Start the clock
	steady_clock::time_point ts, te;
	ts = steady_clock::now();
	fileOut << "\nCPU version\n" << "\nCPU version generating from range (0" << "~" << range << ")\n\n";
	//Keep track of results
	uint64_t count = 0;
	//Outer loop
	for (uint64_t i = 0; i < range; i++)
		//Inner loop
		for (uint64_t j = 2; j*j <= i; j++) {
			if (i % j == 0)
				break;
			else if (j + 1 > sqrt(i)) {
				//User wants to see output on screen
				if (mode) {
					std::cout << std::fixed << i << "\t";
					fileOut << std::fixed << i << "\t";
					count++;
				}
				//Just write to file if mode is 0
				else
				{
					fileOut << std::fixed << i << "\t";
					count++;
				}
			}
		}
	//Stop the clock
	te = steady_clock::now();

	std::cout << "\n\nTotal number of primes: " << count << std::endl;
	reportTime("\nCPU Program Completed in ", te - ts);

	fileOut << "\n\nTotal number of primes: " << count << std::endl;

	std::cout << "A log file with the current date/time has been placed in the program directory.\n";
	std::cout << "--------------------------------------------------------------------------------\n";
}

std::ofstream fileInit(){
	//Get current date and time
	time_t rawtime;
	struct tm * timeinfo;
	char buffer[80];
	time(&rawtime);
	timeinfo = localtime(&rawtime);

	//Format in Year-Month-Day_Hour_Minute_Seconds
	strftime(buffer, 80, "%y-%m-%d_%H-%M-%S", timeinfo);
	std::string dateTime(buffer);

	//File handles
	std::ofstream fileOut;
	fileOut.open("GenPrime_out_" + dateTime + ".txt");
	return fileOut;
}

int setupRange(int range) {
	if (range == 0) {
		std::cout << "[2/3] Please choose the range(3 ~ 500,000): \n";
		std::cin >> range;

		//Error checking
		if (range > 2 && range <= 500000) {
			return range;
		}
		else {
			std::cout << "Invalid input for range, value set to default 500,000\n";
			return 500000;
		}
	}
	else return range;
}

//Array of MAXSIZE is created and filled with prime numbers, where [i]
//is the prime int and the rest is padded with 0's
//Example: cpudata[i] = {0,1,0,3,0,5,0,7,0,0,0,11,0,0,0...}
void justDoIt(int range, bool mode, std::ofstream& fileOut) {
	//Output to file
	fileOut << "CUDA Multithreading Sieve of Eratosthenes\n" << "CUDA Multithreading generating from range (0" << "~" << range << ")\n\n";

	//Filter out even numbers to simplify calculation
	CUDAFilter(data, (range / 2) + 1);

	//Initialize arrays
	int *gpudata;
	int cpudata[MAXSIZE];

	//Allocate memory
	hipMalloc((void**)&gpudata, sizeof(int)*MAXSIZE);

	//Copy to GPU
	hipMemcpy(gpudata, data, sizeof(int)*MAXSIZE, hipMemcpyHostToDevice);

	//Maximum threads per block for CUDA 5.2 is 1024
	int bNum = 96, tNum = 1024;
	
	//Start the clock
	steady_clock::time_point ts, te;
	ts = steady_clock::now();

	//Kernel call on the GPU
	CUDASieve << <bNum, tNum, 0 >> >(gpudata, range, bNum, tNum);
	
	//Synchronize the device and the host
	hipDeviceSynchronize();

	//Copy from GPU back onto host
	hipMemcpy(&cpudata, gpudata, sizeof(int)*MAXSIZE, hipMemcpyDeviceToHost);

	//Free the memory on the GPU
	hipFree(gpudata);

	//Reset the device for easy profiling
	hipDeviceReset();

	//Stop the clock
	te = steady_clock::now();

	//Display on screen
	if (mode == 1) {
		for (int i = 0; i < MAXSIZE; i++) {
			if (cpudata[i] != 0)
				printf("%d\t", cpudata[i]);
		}
	}
	//Count number of primes
	int count = std::count_if(cpudata, cpudata + MAXSIZE, [](int i){ return i; });
	std::cout << "\n\nTotal number of primes: " << count-2 << std::endl;
	
	//Write to file
	for (int i = 0; i < MAXSIZE; i++) {
		if (cpudata[i] != 0) {
			fileOut << cpudata[i] << "\t";
		}
	}
	//Show the amount of time 
	reportTime("GPU Program Completed in ", te - ts);
	fileOut << "\n\nTotal number of primes: " << count - 2 << std::endl;
	std::cout << "A log file with the current date/time has been placed in the program directory.\n";
	std::cout << "--------------------------------------------------------------------------------\n";
}

void menu(int range, bool mode, std::ofstream& fileOut){
	std::cout << "[3/3] Please select the version of the program you want to run\n"
		<< "1. [*****]  CUDA Multithreading Sieve of Eratosthenes version\n"
		<< "2. [***]    Simple CPU version\n"
		<< "3. [**]	Run both versions\n"
		<< "0. Quit\n"
		<< "Option: ";
	int mainMenuOption;
	std::cin >> mainMenuOption;	//Accept user input
		switch (mainMenuOption) {
		case 0:	// User wants to exit
			std::cout << "Thank you for testing our program :)\n"
				<< "Fork us @ https://github.com/bbershadsky/" << std::endl;
			break;
		case 1:
			std::cout << "CUDA Multithreading generating from range (0" << "~" << range << ")\n";
			std::cout << "--------------------------------------------------------------------------------\n";
			justDoIt(range, mode, fileOut);

			//Close the file handle
			fileOut.close();
			break;
		case 2:
			std::cout << "CPU version generating from range (0" << "~" << range << ")\n";
			std::cout << "--------------------------------------------------------------------------------\n";
			CPUgenPrime(range, mode, fileOut);

			//Close the file handle
			fileOut.close();
			break;
		case 3:
			std::cout << "Running all available options\n";
			justDoIt(range, mode, fileOut);
			CPUgenPrime(range, mode, fileOut);

			//Close the file handle
			fileOut.close();
			break;
		default:
			std::cout << "[Invalid option. Only integers 0-3 are allowed]\n";
			menu(range, mode, fileOut);
			break;
		}
}

void setupScreenMode(int range) {
	std::cout << "***Team /dev/null GPU610 PRIME NUMBER GENERATOR v3.5***\n"
		<< "[1/3] Would you like to see the output on screen?\n"
		<< "0 = NO, write to file only\n"
		<< "1 = YES, display on screen\n"
		<< "Show on screen?: ";
		int mode = 1;
		std::cin >> mode;

		//Initialize file handle
		std::ofstream fileOut = fileInit();

		if (mode == 0) {
			std::cout << "***Writing output to file only***\n\n";
			range = setupRange(range);
			menu(range, mode, fileOut);
		}

		else if (mode == 1) {
			std::cout << "***Outputting results on screen***\n\n";
			range = setupRange(range);
			menu(range, mode, fileOut);
		}
		else {
			std::cout << "[Invalid option selected, default option 0 (output to screen) selected]\n\n";
			range = setupRange(range);
			menu(range, 1, fileOut);
		}
}

//Initialize value to be used in the program using command line arguments
int initRuntimeValue(int argc, char* argv[]){
	//Save runtime parameter into local variable, if provided
	int range = 500000;
	if (argc == 1) {
		std::cout << "[No command line parameters provided]\n\n";
		return 0;
	}
	if (argc == 2)
		range = std::atoi(argv[1]);
	if (range > 2 && range < 500000)
		return range;
	else {
		std::cout << "[Bad input for range parameter (must be <= 500,000)]\n"
			<< "Range has been set to 500,000\n";
		return range = 500000;
	}
}

int main(int argc, char* argv[]) {
	//Grab the command line arguments
	int range = initRuntimeValue(argc, argv);

	//Prompt user for mode (verbose or silent)
	setupScreenMode(range);
	std::cout << "Thank you for testing our program :)\n"
		<< "Fork us @ https://github.com/bbershadsky/" << std::endl;
	return 0;
}

/*
CHANGELOG
v1.0 - Generating from simple double loop
v1.0.1 - Command line parameter input
v1.1 - Nicer output format and error feedback
v1.2 - Full 64 bit integer compatibility
v1.3 - Multithreading and CUDA implemented
v2.0 - Completely rewrote program to include menu and multiple run parameters
v3.0 - Full rewrite of CUDAGenPrime to use CUDASieve of Eratosthenes, and initRuntimeValues
v3.1 - Moved new CUDAGenPrime to separate function justDoIt(range);
v3.2 - Reorganized main() into simpler blocks for easier readability and efficiency
v3.3 - Moved most control blocks over to the menu() for easier modification
v3.3.1 - Removed a bunch of unused includes
v3.4 - Successfully fixed file output and implemented count
v3.5 - Final version with usability and performance upgrades
*/
